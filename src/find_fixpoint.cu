#include "hip/hip_runtime.h"
#include <cstdio>

#include <hip/hip_runtime.h>

#include "find_fixpoint.cuh"

CUDA_CALLABLE
uint32_t leftrotate(uint32_t a, uint32_t b) {
    uint32_t high = a << b;
    uint32_t low  = a >> (32 - b);
    return high | low;
}

/* adapted from https://en.wikipedia.org/wiki/SHA-1#SHA-1_pseudocode
   assumes message is PREFIX_LEN bytes
*/
CUDA_CALLABLE
void sha1ofPrefix(uint8_t* result, uint8_t* prefix) {
    uint32_t
        h0 = 0x67452301,
        h1 = 0xEFCDAB89,
        h2 = 0x98BADCFE,
        h3 = 0x10325476,
        h4 = 0xC3D2E1F0;


    uint64_t m1 = 8 * PREFIX_LEN;

    // ASSUMPTION: 0 < PREFIX_LEN < 56
    uint8_t message_padded[64];
    memcpy(message_padded, prefix, PREFIX_LEN);
    message_padded[PREFIX_LEN] = 0x80;

    for (int i = 0; i < 8; i++) {
        message_padded[56 + i] = ((uint8_t*)&m1)[ 7 - i ];
    }

    uint32_t w[80];
    for (int i = 0; i < 16; i++) {
        uint8_t* current_word = (uint8_t*)(w + i);
        for (int byte = 0; byte < 4; byte++) {
            current_word[3 - byte] = message_padded[ chunk + (4 * i) + byte ];
        }
    }

    for (int i = 16; i < 80; i++) {
        w[i] = leftrotate(w[i-3] ^ w[i-8] ^ w[i-14] ^ w[i-16], 1);
    }

    uint32_t a = h0, b = h1, c = h2, d = h3, e = h4, f, k;
    for (int i = 0; i < 80; i++) {
        if (i < 20) {
            f = (b & c) | ((~b) & d);
            k = 0x5A827999;
        } else if (i < 40) {
            f = b ^ c ^ d;
            k = 0x6ED9EBA1;
        } else if (i < 60) {
            f = (b & c) | (b & d) | (c & d);
            k = 0x8F1BBCDC;
        } else {
            f = b ^ c ^ d;
            k = 0xCA62C1D6;
        }

        uint32_t temp = leftrotate(a, 5) + f + e + k + w[i];
        e = d;
        d = c;
        c = leftrotate(b, 30);
        b = a;
        a = temp;
    }

    h0 += a;
    h1 += b;
    h2 += c;
    h3 += d;
    h4 += e;

    for (int i = 0; i < 4; i++) {
        result[ 3 - i] = h0 >> 8 * i;
        result[ 7 - i] = h1 >> 8 * i;
        result[11 - i] = h2 >> 8 * i;
        result[15 - i] = h3 >> 8 * i;
        result[19 - i] = h4 >> 8 * i;
    }
}

__global__
void cudaShaFixpointSearchKernel(bool* success, uint8_t* prefix) {
    PrefixCounter p;
    p.n = blockDim.x * blockIdx.x + threadIdx.x;
    uint8_t result[PREFIX_LEN];

    while (p.n <= PREFIX_COUNTER_MAX) {
        sha1ofPrefix(result, p.prefix);

        if (! memcmp(result, p.prefix, PREFIX_LEN)) {
            *success = true;
            *prefix = p.prefix;
            // TODO quit all threads
        }

        p.n += blockDim.x * gridDim.x;
    }
}


bool cudaCallShaFixpointSearchKernel(const unsigned int blocks,
    const unsigned int threads_per_block, uint8_t* resultDest) {

    bool* success;
    hipMalloc(&success, sizeof(bool));

    uint8_t* prefix;
    hipMalloc(&prefix, PREFIX_LEN * sizeof(uint8_t));

    cudaShaFixpointSearchKernel<<<blocks, threads_per_block>>>(success, prefix);

    bool host_success;
    hipMemcpy(&host_success, success, hipMemcpyDeviceToHost);
    hipMemcpy(&resultDest, prefix, hipMemcpyDeviceToHost);

    return host_success;
}


// TODO
// __global__ void cudaTreeFixpointSearchKernel
// void cudaCall...
